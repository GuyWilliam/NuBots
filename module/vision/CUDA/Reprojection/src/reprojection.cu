#include "hip/hip_runtime.h"
#include "reprojection.cuh"

// Define our texture.
texture<unsigned char, hipTextureType2D, hipReadModeNormalizedFloat> input_image;

__constant__ const unsigned int FORMAT_GREY = 0x59455247;
__constant__ const unsigned int FORMAT_Y12  = 0x20323159;
__constant__ const unsigned int FORMAT_Y16  = 0x20363159;
__constant__ const unsigned int FORMAT_Y411 = 0x31313459;
__constant__ const unsigned int FORMAT_UYVY = 0x59565955;
__constant__ const unsigned int FORMAT_YUYV = 0x56595559;
__constant__ const unsigned int FORMAT_YM24 = 0x34324d59;
__constant__ const unsigned int FORMAT_RGB3 = 0x33424752;
__constant__ const unsigned int FORMAT_JPEG = 0x4745504a;

// bayer formats
__constant__ const unsigned int FORMAT_GRBG = 0x47425247;
__constant__ const unsigned int FORMAT_RGGB = 0x42474752;
__constant__ const unsigned int FORMAT_GBRG = 0x47524247;
__constant__ const unsigned int FORMAT_BGGR = 0x52474742;
__constant__ const unsigned int FORMAT_GR12 = 0x32315247;
__constant__ const unsigned int FORMAT_RG12 = 0x32314752;
__constant__ const unsigned int FORMAT_GB12 = 0x32314247;
__constant__ const unsigned int FORMAT_BG12 = 0x32314742;
__constant__ const unsigned int FORMAT_GR16 = 0x36315247;
__constant__ const unsigned int FORMAT_RG16 = 0x36314752;
__constant__ const unsigned int FORMAT_GB16 = 0x36314247;
__constant__ const unsigned int FORMAT_BG16 = 0x36314742;

/**
 * A function for converting a YCbCr colour to RGBA
 * Based from http://en.wikipedia.org/wiki/YCbCr#JPEG_conversion
 *
 * @param {float4} ycbcr A 4-component YCbCr array (includes alpha for convenience)
 * @returns {float4} A converted RGBA colour (alpha untouched)
 */
__device__ float4 YCbCrToRGB(float4 ycbcr) {
    const float factor = 128.0f / 255.0f;

    // conversion numbers have been modified to account for the colour being in the 0-1 range instead of 0-255
    return make_float4(min(max(ycbcr.x + 1.402f * (ycbcr.z - factor), 0.0f), 1.0f),
                       min(max(ycbcr.x - 0.34414f * (ycbcr.y - factor) - 0.71414f * (ycbcr.z - factor), 0.0f), 1.0f),
                       min(max(ycbcr.x + 1.772f * (ycbcr.y - factor), 0.0f), 1.0f),
                       min(max(ycbcr.w, 0.0f), 1.0f));
}

// http://graphics.cs.williams.edu/papers/BayerJGT09/
__device__ float4 bayerToRGB(float2 sample_point, float2 first_red) {
    float4 center = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    center.x      = sample_point.x;
    center.y      = sample_point.y;
    center.z      = sample_point.x + first_red.x;
    center.w      = sample_point.y + first_red.y;

    float4 x_coord = center.x + make_float4(-2.0f, -1.0f, 1.0f, 2.0f);
    float4 y_coord = center.y + make_float4(-2.0f, -1.0f, 1.0f, 2.0f);

    float C         = tex2D(input_image, center.x, center.y);  // (0, 0)
    const float4 kC = make_float4(0.5f, 0.75f, 0.625f, 0.625f);

    // Determine which of four types of pixels we are on.
    float2 alternate = make_float2(fmod(floor(center.z), 2.0f), fmod(floor(center.w), 2.0f));

    float4 Dvec = make_float4(tex2D(input_image, x_coord.y, y_coord.y),   // (-1, -1)
                              tex2D(input_image, x_coord.y, y_coord.z),   // (-1,  1)
                              tex2D(input_image, x_coord.z, y_coord.y),   // ( 1, -1)
                              tex2D(input_image, x_coord.z, y_coord.z));  // ( 1,  1)

    const float3 kC_temp = make_float3(kC.x * C, kC.y * C, kC.z * C);
    float4 PATTERN       = make_float4(kC_temp.x, kC_temp.y, kC_temp.z, kC_temp.z);

    // Can also be a dot product with (1,1,1,1) on hardware where that is
    // specially optimized.
    // Equivalent to: D = Dvec.x + Dvec.y + Dvec.z + Dvec.w;
    Dvec.x += Dvec.z;
    Dvec.y += Dvec.w;
    Dvec.x += Dvec.y;

    float4 value = make_float4(tex2D(input_image, center.x, y_coord.x),   // ( 0, -2)
                               tex2D(input_image, center.x, y_coord.y),   // ( 0, -1)
                               tex2D(input_image, x_coord.x, center.y),   // (-1,  0)
                               tex2D(input_image, x_coord.y, center.y));  // (-2,  0)

    float4 temp = make_float4(tex2D(input_image, center.x, y_coord.w),   // (0, 2)
                              tex2D(input_image, center.x, y_coord.z),   // (0, 1)
                              tex2D(input_image, x_coord.w, center.y),   // (2, 0)
                              tex2D(input_image, x_coord.z, center.y));  // (1, 0)

    // Even the simplest compilers should be able to constant-fold these to avoid the division.
    // Note that on scalar processors these constants force computation of some identical products twice.
    const float4 kA = make_float4(-0.125f, -0.1875f, 0.0625f, -0.125f);
    const float4 kB = make_float4(0.25f, 0.0f, 0.0f, 0.5f);
    const float4 kD = make_float4(0.0f, 0.25f, -0.125f, -0.125f);

    // Conserve constant registers and take advantage of free swizzle on load
    const float4 kE = make_float4(kA.x, kA.y, kA.w, kA.z);
    const float4 kF = make_float4(kB.x, kB.y, kB.w, kB.z);

    value += temp;

    // There are five filter patterns (identity, cross, checker,
    // theta, phi).  Precompute the terms from all of them and then
    // use swizzles to assign to color channels.
    //
    // Channel   Matches
    //   x       cross   (e.g., EE G)
    //   y       checker (e.g., EE B)
    //   z       theta   (e.g., EO R)
    //   w       phi     (e.g., EO R)
    const float A = value.x;
    const float B = value.y;
    const float D = Dvec.x;
    const float E = value.z;
    const float F = value.w;

    // Avoid zero elements. On a scalar processor this saves two MADDs and it has no
    // effect on a vector processor.
    const float2 kD_temp = make_float2(kD.y * D, kD.z * D);
    PATTERN.y += kD_temp.x;
    PATTERN.z += kD_temp.y;
    PATTERN.w += kD_temp.y;

    const float3 kA_temp = make_float3(kA.x * A, kA.y * A, kA.z * A);
    const float3 kE_temp = make_float3(kE.x * E, kE.y * E, kE.w * E);
    PATTERN += make_float4(kA_temp.x, kA_temp.y, kA_temp.z, kA_temp.x);
    PATTERN += make_float4(kE_temp.x, kE_temp.y, kE_temp.x, kE_temp.z);
    PATTERN.x += kB.x * B;
    PATTERN.w += kB.w * B;
    PATTERN.x += kF.x * F;
    PATTERN.z += kF.z * F;

    float4 result;

    if (alternate.y == 0.0f) {
        if (alternate.x == 0.0f) {
            result = make_float4(C, PATTERN.x, PATTERN.y, 1.0f);
        }

        else {
            result = make_float4(PATTERN.z, C, PATTERN.w, 1.0f);
        }
    }

    else {
        if (alternate.x == 0.0f) {
            result = make_float4(PATTERN.w, C, PATTERN.z, 1.0f);
        }

        else {
            result = make_float4(PATTERN.y, PATTERN.x, C, 1.0f);
        }
    }

    return result;
}

__device__ float2 projectCamSpaceToScreen(float3 point, float radians_per_pixel) {
    float theta = acosf(point.x);

    if (theta == 0.0f) {
        return make_float2(0.0, 0.0);
    }

    float r         = theta / radians_per_pixel;
    float sin_theta = sinf(theta);
    float px        = r * point.y / (sin_theta);
    float py        = r * point.z / (sin_theta);

    return make_float2(px, py);
}

__device__ float3 getCamFromScreen(float2 screen, float cam_focal_length_pixels) {
    return normalize(make_float3(cam_focal_length_pixels, screen.x, screen.y));
}

__global__ void projectSphericalToRectilinear(unsigned int image_format,
                                              float radians_per_pixel,
                                              uint2 input_dimensions,
                                              uint2 output_dimensions,
                                              float cam_focal_length_pixels,
                                              unsigned char* output) {

    // Calculate input texture coordinates
    const uint2 pos = make_uint2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    const float2 output_center = make_float2((output_dimensions.x - 1.0f) * 0.5f, (output_dimensions.y - 1.0) * 0.5);
    const float2 input_center  = make_float2((input_dimensions.x - 1.0f) * 0.5f, (input_dimensions.y - 1.0) * 0.5);

    float2 centered_point = make_float2(output_center.x - pos.x, output_center.y - pos.y);
    float2 projected_point =
        projectCamSpaceToScreen(getCamFromScreen(centered_point, cam_focal_length_pixels), radians_per_pixel);
    float2 sample_point = make_float2(input_center.x - projected_point.x, input_center.y - projected_point.y);
    float4 colour;

    // convert into RGBA colour
    switch (image_format) {
        case FORMAT_GRBG:
            colour = bayerToRGB(sample_point, make_float2(1.0, 0.0));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 0] =
                __float2uint_rz(clamp(255.0f * colour.x + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 1] =
                __float2uint_rz(clamp(255.0f * colour.y + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 2] =
                __float2uint_rz(clamp(255.0f * colour.z + 0.5f, 0.0f, 255.0f));
            break;

        case FORMAT_RGGB:
            colour = bayerToRGB(sample_point, make_float2(0.0, 0.0));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 0] =
                __float2uint_rz(clamp(255.0f * colour.x + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 1] =
                __float2uint_rz(clamp(255.0f * colour.y + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 2] =
                __float2uint_rz(clamp(255.0f * colour.z + 0.5f, 0.0f, 255.0f));
            break;

        case FORMAT_GBRG:
            colour = bayerToRGB(sample_point, make_float2(0.0, 1.0));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 0] =
                __float2uint_rz(clamp(255.0f * colour.x + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 1] =
                __float2uint_rz(clamp(255.0f * colour.y + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 2] =
                __float2uint_rz(clamp(255.0f * colour.z + 0.5f, 0.0f, 255.0f));
            break;

        case FORMAT_BGGR:
            colour = bayerToRGB(sample_point, make_float2(1.0, 1.0));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 0] =
                __float2uint_rz(clamp(255.0f * colour.x + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 1] =
                __float2uint_rz(clamp(255.0f * colour.y + 0.5f, 0.0f, 255.0f));
            output[pos.y * output_dimensions.x * 3 + pos.x * 3 + 2] =
                __float2uint_rz(clamp(255.0f * colour.z + 0.5f, 0.0f, 255.0f));
            break;

        // We don't handle these.
        case FORMAT_GREY:
        case FORMAT_Y12:
        case FORMAT_Y16:
        case FORMAT_Y411:
        case FORMAT_GR12:
        case FORMAT_RG12:
        case FORMAT_GB12:
        case FORMAT_BG12:
        case FORMAT_GR16:
        case FORMAT_RG16:
        case FORMAT_GB16:
        case FORMAT_BG16:
        case FORMAT_YUYV:
        case FORMAT_YM24:
        case FORMAT_JPEG:
        case FORMAT_UYVY:
        case FORMAT_RGB3:
        default:
            output[pos.y * input_dimensions.x * 3 + pos.x * 3 + 0] = 0;
            output[pos.y * input_dimensions.x * 3 + pos.x * 3 + 1] = 0;
            output[pos.y * input_dimensions.x * 3 + pos.x * 3 + 2] = 0;
            break;
    }
}

hipError_t launchKernel(const unsigned char* input,
                         unsigned int image_format,
                         float radians_per_pixel,
                         uint2 input_dimensions,
                         uint2 output_dimensions,
                         float cam_focal_length_pixels,
                         unsigned char* output) {

    hipError_t err;

    // Set texture reference parameters
    input_image.addressMode[0] = hipAddressModeClamp;  // Clamp to edge
    input_image.addressMode[1] = hipAddressModeClamp;
    input_image.filterMode     = hipFilterModePoint;  // No interpolation
    input_image.normalized     = false;                // Texture coordinates are not normalised.

    // Allocate device memory for texture.
    unsigned char* texture;
    size_t pitch;
    if ((err = hipMallocPitch(&texture, &pitch, input_dimensions.x, input_dimensions.y)) != hipSuccess) {
        return err;
    }

    // Copy texture to device.
    if ((hipMemcpy2D(
            texture, pitch, input, input_dimensions.x, input_dimensions.x, input_dimensions.y, hipMemcpyHostToDevice))
        != hipSuccess) {
        hipFree(texture);
        return err;
    }

    // Bind texture to device memory.
    size_t offset;
    if ((hipBindTexture2D(&offset,
                           input_image,
                           texture,
                           hipCreateChannelDesc<unsigned char>(),
                           input_dimensions.x,
                           input_dimensions.y,
                           pitch))
        != hipSuccess) {
        hipFree(texture);
        return err;
    }

    // Allocate memory for kernel output.
    unsigned char* result;
    if ((hipMalloc(&result, output_dimensions.x * output_dimensions.y * 3 * sizeof(unsigned char))) != hipSuccess) {
        hipUnbindTexture(input_image);
        hipFree(texture);
        return err;
    }

    // Set up kernel execution parameters.
    dim3 dimBlock(16, 16);
    dim3 dimGrid((output_dimensions.x + dimBlock.x - 1) / dimBlock.x,
                 (output_dimensions.y + dimBlock.y - 1) / dimBlock.y);

    // Execute the kernel.
    projectSphericalToRectilinear<<<dimGrid, dimBlock>>>(
        image_format, radians_per_pixel, input_dimensions, output_dimensions, cam_focal_length_pixels, result);

    // Check for any errors.
    if ((err = hipGetLastError()) != hipSuccess) {
        hipUnbindTexture(input_image);
        hipFree(texture);
        hipFree(output);
        return err;
    }

    // Copy the result out of the device.
    if ((err = hipMemcpy(output, result, output_dimensions.x * output_dimensions.y * 3, hipMemcpyDeviceToHost))
        != hipSuccess) {
        hipUnbindTexture(input_image);
        hipFree(texture);
        hipFree(output);
        return err;
    }

    // Clean up.
    hipUnbindTexture(input_image);
    hipFree(texture);
    hipFree(result);

    return hipSuccess;
}
